#include "hip/hip_runtime.h"
/*
 * (C) Copyright 1996-2017 ECMWF.
 *
 * This software is licensed under the terms of the Apache Licence Version 2.0
 * which can be obtained at http://www.apache.org/licenses/LICENSE-2.0.
 * In applying this licence, ECMWF does not waive the privileges and immunities
 * granted to it by virtue of its status as an intergovernmental organisation nor
 * does it submit to any jurisdiction.
 */

#include "HaloExchangeCUDA.h"

namespace atlas {
namespace parallel {

template<typename DATA_TYPE, int RANK>
__global__ void pack_kernel(const int sendcnt, const array::SVector<int> sendmap,
         const array::ArrayView<DATA_TYPE, RANK, false> field, array::SVector<DATA_TYPE> send_buffer,
                            typename std::enable_if<RANK==2, int>::type* = 0) {
    const size_t p = blockIdx.x*blockDim.x + threadIdx.x;
    const size_t i = blockIdx.y*blockDim.y + threadIdx.y;

    if(p >= sendcnt || i >= field.data_view().template length<1>() ) return;

    const size_t buff_idx = field.data_view().template length<1>() * p + i;

    send_buffer[buff_idx] = field(sendmap[p], i);
}

template<typename DATA_TYPE, int RANK>
__global__ void pack_kernel(const int sendcnt, const array::SVector<int> sendmap,
         const array::ArrayView<DATA_TYPE, RANK, false> field, array::SVector<DATA_TYPE> send_buffer,
                            typename std::enable_if<RANK!=2, int>::type* = 0) {
}

template<typename DATA_TYPE, int RANK>
__global__ void unpack_kernel(const int sendcnt, const array::SVector<int> recvmap,
         const array::SVector<DATA_TYPE> recv_buffer, array::ArrayView<DATA_TYPE, RANK> field,
                            typename std::enable_if<RANK==2, int>::type* = 0) {

    const size_t p = blockIdx.x*blockDim.x + threadIdx.x;
    const size_t i = blockIdx.y*blockDim.y + threadIdx.y;

    if(p >= sendcnt || i >= field.data_view().template length<1>() ) return;

    const size_t buff_idx = field.data_view().template length<1>() * p + i;

    field(recvmap[p], i) = recv_buffer[buff_idx];
}

template<typename DATA_TYPE, int RANK>
__global__ void unpack_kernel(const int sendcnt, const array::SVector<int> recvmap,
         const array::SVector<DATA_TYPE> recv_buffer, array::ArrayView<DATA_TYPE, RANK> field,
                            typename std::enable_if<RANK!=2, int>::type* = 0) {
}

template<typename DATA_TYPE>
void halo_packer_cuda<DATA_TYPE, 1>::pack( const int sendcnt, array::SVector<int> const & sendmap,
                   const array::ArrayView<DATA_TYPE, 1>& field, array::SVector<DATA_TYPE>& send_buffer )
{
}

template<typename DATA_TYPE>
void halo_packer_cuda<DATA_TYPE, 1>::unpack(const int sendcnt, array::SVector<int> const & recvmap,
                   const array::SVector<DATA_TYPE> &recv_buffer ,
                   array::ArrayView<DATA_TYPE, 1> &field)
{
}

template<typename DATA_TYPE, int RANK>
void halo_packer_cuda<DATA_TYPE, RANK>::pack( const int sendcnt, array::SVector<int> const & sendmap,
                   const array::ArrayView<DATA_TYPE, RANK>& field, array::SVector<DATA_TYPE>& send_buffer )
{
  const unsigned int block_size_x = 32;
  const unsigned int block_size_y = 4;
  dim3 threads(block_size_x, block_size_y);
  dim3 blocks((sendcnt+block_size_x-1)/block_size_x, (field.data_view().template length<1>()+block_size_y-1)/block_size_y);

  pack_kernel<DATA_TYPE, RANK><<<blocks,threads>>>(sendcnt, sendmap, field, send_buffer);
}

template<typename DATA_TYPE, int RANK>
void halo_packer_cuda<DATA_TYPE, RANK>::unpack(const int sendcnt, array::SVector<int> const & recvmap,
                   const array::SVector<DATA_TYPE> &recv_buffer ,
                   array::ArrayView<DATA_TYPE, RANK> &field)
{
  const unsigned int block_size_x = 32;
  const unsigned int block_size_y = 4;
  dim3 threads(block_size_x, block_size_y);
  dim3 blocks((sendcnt+block_size_x-1)/block_size_x, (field.data_view().template length<1>()+block_size_y-1)/block_size_y);

  unpack_kernel<<<blocks,threads>>>(sendcnt, recvmap, recv_buffer, field);
}

#define EXPLICIT_TEMPLATE_INSTANTIATION(RANK) \
template class halo_packer_cuda<int,RANK>; \
template class halo_packer_cuda<long,RANK>; \
template class halo_packer_cuda<long unsigned,RANK>; \
template class halo_packer_cuda<float,RANK>; \
template class halo_packer_cuda<double,RANK>; \

  EXPLICIT_TEMPLATE_INSTANTIATION(1)
  EXPLICIT_TEMPLATE_INSTANTIATION(2)
  EXPLICIT_TEMPLATE_INSTANTIATION(3)
  EXPLICIT_TEMPLATE_INSTANTIATION(4)

} //namespace array
} //namespace atlas
