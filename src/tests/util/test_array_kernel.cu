#include "hip/hip_runtime.h"
/*
 * (C) Copyright 1996-2016 ECMWF.
 *
 * This software is licensed under the terms of the Apache Licence Version 2.0
 * which can be obtained at http://www.apache.org/licenses/LICENSE-2.0.
 * In applying this licence, ECMWF does not waive the privileges and immunities
 * granted to it by virtue of its status as an intergovernmental organisation nor
 * does it submit to any jurisdiction.
 */

#include <hip/hip_runtime.h>
#include "eckit/testing/Test.h"
#include "atlas/array.h"
#include "atlas/array/MakeView.h"
#include "atlas/runtime/Log.h"

using namespace atlas::array;

namespace atlas {
namespace test {

template<typename Value, int RANK>
__global__
void kernel_ex(ArrayView<Value, RANK> dv)
{
    dv(3, 3, 3) += 1;
}

CASE( "test_array" )
{
   Array* ds = Array::create<double>(4ul, 4ul, 4ul);
   ArrayView<double,3> hv = make_host_view<double, 3>(*ds);
   hv(3, 3, 3) = 4.5;

   ds->cloneToDevice();

   auto cv = make_device_view<double, 3>(*ds);

   kernel_ex<<<1,1>>>(cv);

   hipDeviceSynchronize();

   ds->cloneFromDevice();
   ds->reactivateHostWriteViews();

   BOOST_CHECK_EQUAL( hv(3, 3, 3) , 5.5 );

   delete ds;
}

}
}

int main(int argc, char **argv) {
    atlas::test::AtlasTestEnvironment env( argc, argv );
    return run_tests ( argc, argv, false );
}
