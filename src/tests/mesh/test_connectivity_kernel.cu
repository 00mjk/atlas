#include "hip/hip_runtime.h"
/*
 * (C) Copyright 1996-2016 ECMWF.
 *
 * This software is licensed under the terms of the Apache Licence Version 2.0
 * which can be obtained at http://www.apache.org/licenses/LICENSE-2.0.
 * In applying this licence, ECMWF does not waive the privileges and immunities
 * granted to it by virtue of its status as an intergovernmental organisation nor
 * does it submit to any jurisdiction.
 */

#define BOOST_TEST_MODULE TestConnectivityKernel
#include <hip/hip_runtime.h>
#include "ecbuild/boost_test_framework.h"
#include "atlas/mesh/Connectivity.h"

using namespace atlas::mesh;

namespace atlas {
namespace test {

#ifdef ATLAS_HAVE_FORTRAN
#define FORTRAN_BASE 1
#define INDEX_REF Index
#define FROM_FORTRAN -1
#define TO_FORTRAN   +1
#else
#define FORTRAN_BASE 0
#define INDEX_REF *
#define FROM_FORTRAN
#define TO_FORTRAN
#endif


__global__
void kernel_ex(BlockConnectivity* conn, bool* result)
{

    *result &= (conn->rows() == 2);
    *result &= (conn->cols() == 5);

    *result &= ((*conn)(0,2) == 9 + FROM_FORTRAN+FORTRAN_BASE);
    *result &= ((*conn)(0,4) == 356 + FROM_FORTRAN+FORTRAN_BASE);
    *result &= ((*conn)(1,1) == 3 + FROM_FORTRAN+FORTRAN_BASE);

}

BOOST_AUTO_TEST_CASE( test_connectivity )
{
    BlockConnectivity* conn;
    hipMallocManaged(&conn, sizeof(BlockConnectivity));

    bool* result;
    hipMallocManaged(&result, sizeof(bool));

    *result = true;

    idx_t vals2[12] = {2,3,9,34,356,86,3,24,84,45,2,2};

    conn->add(2,5, vals2);


//    Array* ds = Array::create<double>(4ul, 4ul, 4ul);
//   ArrayView<double,3> hv = make_host_view<double, 3>(*ds);
//   hv(3, 3, 3) = 4.5;


//   ds->clone_to_device();


//   auto cv = make_device_view<double, 3>(*ds);

   kernel_ex<<<1,1>>>(conn, result);

   hipDeviceSynchronize();

   BOOST_CHECK_EQUAL( *result , true );


//   ds->clone_from_device();
//   ds->reactivate_host_write_views();

//   BOOST_CHECK_EQUAL( hv(3, 3, 3) , 5.5 );

//   delete ds;
}

}
}
